#include "hip/hip_runtime.h"
#include <cmath>
#include <torch/extension.h>


__global__ void forward_kernel(float *Q, float *K, float *V, float *O, float *l, float *m, const int N, const int d,
                               const int Bc, const int Br, const int Tc, const int Tr, const float scale, 
                               const int32_t* mf, const int dm) {
    // Given Q, K, V, we need to compute O
    //
    // Q, K, V: query, key, value (N * d)
    // O, output: (N * d)
    // l, m: intermediate states (N)
    // N: sequence length <int>(scaler)
    // d: dimention <int>(scaler)
    // Bc, Br: number of col/row per block
    // Tc, Tr: number of blocks

    int batch_id = blockIdx.x;
    int head_id = blockIdx.y;
    int thread_id = threadIdx.x;
    int num_head = gridDim.y;
    int num_threads = blockDim.x;  // num_threads=Br

    // Differnt offset for different (batch, head)
    int qkv_offset = (batch_id * num_head * N * d) + (head_id * N * d);
    int mask_offset = batch_id * N * dm;
    int lm_offset = (batch_id * num_head * N) + (head_id * N);

    // Shared memory stored K, V, Q, SP;
    // Note: SP stored Sij and Pij
    // Note: Why SPij in shared memory? because their size is dynamic
    extern __shared__ float sram[];
    float *const smem_Kj = &sram[0];                                              // size=(Bc * d)
    float *const smem_Vj = &sram[Bc * d];                                         // size=(Bc * d)
    float *const smem_Qi = &sram[Bc * d + Bc * d];                                // size=(Br * d)
    float *const smem_SPij = &sram[Bc * d + Bc * d + Br * d];                     // size=(Bc * Br)
    int32_t *const smem_Mi = reinterpret_cast<int32_t *const>(&sram[Bc * d + Bc * d + Br * d + Bc * Br]);           // size=(Br * dm)
    int32_t *const smem_Mj = reinterpret_cast<int32_t *const>(&sram[Bc * d + Bc * d + Br * d + Bc * Br + Br * dm]); // size=(Bc * dm)

    const int offset_si = thread_id * Bc;  // Different thread process different row of Sij

    for (int j = 0; j < Tc; ++j) {
        // Note: Each thread may load multiple columns, since Bc != Br.
        for (int y = thread_id; y < Bc; y += num_threads) {
            int global_col = j * Bc + y;
            if (global_col < N) {  // Make sure global col < seq_len
                // Load Ki, Vj to the shared_memory
                // Kj, Vj: Bc*d
                for (int x = 0; x < d; x += 1) {
                    smem_Kj[y * d + x] = K[qkv_offset + (j * Bc * d) + (y * d) + x];
                    smem_Vj[y * d + x] = V[qkv_offset + (j * Bc * d) + (y * d) + x];
                }

                // load Mj to sram
                for(int x = 0; x < dm; x++) {
                    smem_Mj[y * dm + x] = mf[mask_offset + (j * Bc * dm) + (y * dm) + x];
                }
            }
        }
        __syncthreads();

        const int num_cols = min(Bc, N - (Bc * j));
        for (int i = 0; i < Tr; ++i) {
            int global_row = i * Br + thread_id;

            if (global_row < N) {  // Make sure global row < seq_len
                // Collaboratively Load Qi to shared memory
                for (int x = 0; x < d; x += 1) {
                    smem_Qi[thread_id * d + x] = Q[qkv_offset + (i * Br * d) + (thread_id * d) + x];
                }

                // load Mi to sram
                for (int x = 0; x < dm; x++) {
                    smem_Mi[thread_id * dm + x] = mf[mask_offset + (i * Br * dm) + (thread_id * dm) + x];
                }

                __syncthreads();

                // Load li, mi from HBM to register
                float li = l[lm_offset + (i * Br) + thread_id];
                float mi = m[lm_offset + (i * Br) + thread_id];

                // Compute Sij = Qi * Kj^transpose
                for (int c = 0; c < num_cols; c += 1) {
                    float dot = 0;
                    for (int x = 0; x < d; x += 1) {
                        dot += (smem_Qi[thread_id * d + x] * smem_Kj[c * d + x]);
                    }
                    smem_SPij[offset_si + c] = dot * scale;
                }

                // mask = Mi @ Mj^T, apply mask bias
                for(int c = 0; c < num_cols; c++){
                    int masksum = 0;
                    for(int x = 0; x < dm; x++){
                        masksum += (smem_Mi[thread_id * dm + x] * smem_Mj[c * dm + x]);
                    }

                    smem_SPij[offset_si + c] += masksum > 0 ? 0.0 : -INFINITY;
                }

                // Find new maximum mi for each row
                float mi_tilde = -INFINITY;  // maximum inside this block
                for (int c = 0; c < num_cols; c += 1) {
                    mi_tilde = max(mi_tilde, smem_SPij[offset_si + c]);
                }

                // Calculate Pij & li_tilde
                float li_tilde = 0;
                for (int c = 0; c < num_cols; c += 1) {
                    smem_SPij[offset_si + c] = __expf(smem_SPij[offset_si + c] - mi_tilde);
                    li_tilde += smem_SPij[offset_si + c];
                }

                // Compute mi_new, li_new
                float mi_new = max(mi, mi_tilde);
                float li_new = __expf(mi - mi_new) * li + __expf(mi_tilde - mi_new) * li_tilde;

                // Write Oi to HBM
                for (int x = 0; x < d; x += 1) {
                    // Calculate Pij * Vj
                    float pv_dot = 0;
                    for (int c = 0; c < num_cols; c += 1) {
                        pv_dot += smem_SPij[offset_si + c] * smem_Vj[c * d + x];
                    }

                    O[qkv_offset + (i * Br * d) + (thread_id * d) + x] =
                        (1 / li_new) *
                        ((li * __expf(mi - mi_new) * O[qkv_offset + (i * Br * d) + (thread_id * d) + x]) +
                         __expf(mi_tilde - mi_new) * pv_dot);
                }

                // Write li, mi to HBM
                l[lm_offset + (i * Br) + thread_id] = li_new;
                m[lm_offset + (i * Br) + thread_id] = mi_new;
            }
            // Make sure both Kj and Vj are correct.
            __syncthreads();
        }
    }
}

inline void CHECK_CUDA_ERROR() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(err);
    }
}

void launch_forward_kernel(torch::Tensor Q, torch::Tensor K, torch::Tensor V, torch::Tensor l, torch::Tensor m,
                           torch::Tensor O, torch::Tensor mf) {
    //
    int batch_size = Q.size(0);
    int num_heads = Q.size(1);
    int N = Q.size(2);
    int d = Q.size(3);
    int dm = mf.size(2);

    printf("Batch=%d, Head=%d, SeqLen=%d, EmbDim=%d\n", batch_size, num_heads, N, d);
    //
    int max_shared_memory;
    int max_threads_num;
    hipDeviceGetAttribute(&max_shared_memory, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    hipDeviceGetAttribute(&max_threads_num, hipDeviceAttributeMaxThreadsPerBlock, 0);
    // 
    int M = max_shared_memory / sizeof(float);  // number of floats shared memory can hold

    // int Bc = std::ceil(M / (4 * d));
    // int Br = std::min(std::min(Bc, d), max_threads_num);

    int Bc = 32;
    int Br = 32;
    printf("Br=%d, Bc=%d\n", Br, Bc);
    
    int Tc = (int)std::ceil(float(N) / Bc);
    int Tr = (int)std::ceil(float(N) / Br);

    dim3 grid_dim(batch_size, num_heads);
    dim3 thread_block_dim(Br);
    // shared_memory_size
    // For: Ki, Vi, Qi, Sij, Mi, Mj
    const int shared_memory_size = sizeof(float) * ((2 * Bc * d) + (Br * d) + (Bc * Br)) + sizeof(int32_t) * ((Bc * dm) + (Br * dm));

    printf("Max_shared(bytes)=%d, Max_shared(#dtype)=%d, Requested_memory(bytes)=%d\n", max_shared_memory, M,
           shared_memory_size);
    TORCH_CHECK(shared_memory_size < max_shared_memory, "Shared memory size exceeds the device limit");
    printf("N=%d, d=%d, Bc=%d, Br=%d, Tc=%d, Tr=%d\n", N, d, Bc, Br, Tc, Tr);

    float scale = 1.0f / std::sqrt(static_cast<float>(K.size(3)));

    // Launch
    forward_kernel<<<grid_dim, thread_block_dim, shared_memory_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(), O.data_ptr<float>(), l.data_ptr<float>(),
        m.data_ptr<float>(), N, d, Bc, Br, Tc, Tr, scale,
        mf.data_ptr<int32_t>(), dm);

    CHECK_CUDA_ERROR();
}
